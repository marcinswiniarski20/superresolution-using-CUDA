#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <cmath>
#include <hipfft/hipfft.h>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

using namespace::cv;
using namespace::std;
using namespace::chrono;

#define M_PI 3.14159265358979323846

__global__ void paddingGPU(hipfftComplex *hshift_signal_fft, hipfftComplex *h_padding, int x_dim, int y_dim, int x_res_dim, int y_res_dim)
{
	int diff_x = x_res_dim - x_dim;
	int diff_y = y_res_dim - y_dim;

	int up_offset_x = (int)ceilf(diff_x / 2);
	int up_offset_y = (int)ceilf(diff_y / 2);

	int down_offset_x = x_res_dim - (x_dim + up_offset_x);
	int down_offset_y = y_res_dim - (y_dim + up_offset_y);

	int i = threadIdx.x + blockDim.x*blockIdx.x + up_offset_x;
	int j = threadIdx.y + blockDim.y*blockIdx.y + up_offset_y;

	if (i < x_res_dim - down_offset_x && j < y_res_dim - down_offset_y)
	{
		h_padding[j + i * x_res_dim].x = hshift_signal_fft[j - up_offset_y + (i - up_offset_x)*x_dim].x;
		h_padding[j + i * x_res_dim].y = hshift_signal_fft[j - up_offset_y + (i - up_offset_x)*x_dim].y;
	}
}

__global__ void circshiftGPU(hipfftComplex *in, hipfftComplex *out, int xdim, int ydim, int xshift, int yshift)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;
	if (i < xdim)
	{
		int x = (i + xshift) % xdim;
		if (j < ydim)
		{
			int y = (j + yshift) % ydim;
			out[y + x * ydim].x = in[j + i * ydim].x;
			out[y + x * ydim].y = in[j + i * ydim].y;
		}
	}
}

Mat calcDFT2CPU(Mat &img)
{
	int width = img.cols;
	int height = img.rows;
	Mat fourierIMG = Mat::zeros(width, height, CV_64F);
	double **realOut = new double*[height];
	double **imagOut = new double*[height];

	double **amplitude = new double*[height];
	for (int i = 0; i < height; i++)
	{
		realOut[i] = new double[width];
		imagOut[i] = new double[width];
		amplitude[i] = new double[width];
	}

	for (int yWave = 0; yWave < height; yWave++)
	{
		for (int xWave = 0; xWave < width; xWave++)
		{
			realOut[yWave][xWave] = 0.0;
			imagOut[yWave][xWave] = 0.0;
			for (int ySpace = 0; ySpace < height; ySpace++)
			{
				for (int xSpace = 0; xSpace < width; xSpace++)
				{
					realOut[yWave][xWave] += (img.at<double>(ySpace, xSpace) * cos(
						2 * M_PI * ((1.0 * xWave * xSpace / width) + (1.0
							* yWave * ySpace / height)))) / sqrt(
								width * height);
					imagOut[yWave][xWave] -= (img.at<double>(ySpace, xSpace) * sin(
						2 * M_PI * ((1.0 * xWave * xSpace / width) + (1.0
							* yWave * ySpace / height)))) / sqrt(
								width * height);
				}
			}
			amplitude[yWave][xWave] = sqrt(
				(realOut[yWave][xWave] * realOut[yWave][xWave])
				+ (imagOut[yWave][xWave]
					* imagOut[yWave][xWave]));
			fourierIMG.at<double>(yWave, xWave) = amplitude[yWave][xWave];
		}
	}

	for (int i = 0; i < height; i++)
	{
		delete[] realOut[i];
		delete[] imagOut[i];
		delete[] amplitude[i];
	}
	delete[] realOut;
	delete[] imagOut;
	delete[] amplitude;

	return fourierIMG;
}

void circshift(hipfftComplex *in, hipfftComplex *out, int xdim, int ydim, int xshift, int yshift)
{
	for (int i = 0; i < xdim; i++) {
		int ii = (i + xshift) % xdim;
		//if (ii < 0) ii = xdim + ii;
		for (int j = 0; j < ydim; j++) {
			int jj = (j + yshift) % ydim;
			//if (jj < 0) jj = ydim + jj;
			out[ii * ydim + jj].x = in[i * ydim + j].x;
			out[ii * ydim + jj].y = in[i * ydim + j].y;
		}
	}

}

void fftshift(hipfftComplex *in, hipfftComplex *out, int xdim, int ydim)
{
	circshift(in, out, xdim, ydim, (xdim / 2), (ydim / 2));
}

void ifftshift(hipfftComplex *in, hipfftComplex *out, int xdim, int ydim)
{
	circshift(in, out, xdim, ydim, ((xdim + 1) / 2), ((ydim + 1) / 2));
}

void padding(hipfftComplex *hshift_signal_fft, hipfftComplex *h_padding, int x_dim, int y_dim, int x_res_dim, int y_res_dim)
{
	int diff_x = x_res_dim - x_dim;
	int diff_y = y_res_dim - y_dim;

	int up_offset_x = (int)ceil(diff_x / 2);
	int up_offset_y = (int)ceil(diff_y / 2);

	int down_offset_x = x_res_dim - (x_dim + up_offset_x);
	int down_offset_y = y_res_dim - (y_dim + up_offset_y);

	int temp = 0;
	for (int i = up_offset_x; i < x_res_dim - down_offset_x; i++)
	{
		for (int j = up_offset_y; j < y_res_dim - down_offset_y; j++)
		{
			h_padding[j + i * y_res_dim].x = hshift_signal_fft[temp].x;
			h_padding[j + i * y_res_dim].y = hshift_signal_fft[temp++].y;
		}
	}
}

void printMatrix(hipfftComplex * mat, int ROW, int COL)
{
	std::cout << "\n Printing Matrix : \n";
	for (int i = 0; i <= ROW - 1; i++) {
		for (int j = 0; j <= COL - 1; j++)
			std::cout << mat[j + i * ROW].x << " ";
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

void calcFFTCPU(hipfftComplex *h_signal, hipfftComplex *h_padding, hipfftComplex *h_reversed_signal, int NX, int NY, int NX_RES, int NY_RES, float &time)
{
	hipfftComplex *d_signal, *d_signal_t;
	hipfftComplex *h_signal_fft = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX * NY);
	hipfftComplex *h_shift_signal_fft = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX * NY);
	hipfftComplex *h_signal_ifft = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX_RES * NY_RES);
	hipfftComplex *h_shift_signal_ifft = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX_RES * NY_RES);

	hipMalloc((void **)&d_signal, NX * NY * sizeof(hipfftComplex));
	hipMalloc((void**)&d_signal_t, NX_RES*NY_RES * sizeof(hipfftComplex));
	hipSetDevice(0);
	hipMemcpy(d_signal, h_signal, NX * NY * sizeof(hipfftComplex), hipMemcpyHostToDevice);

	printf("Transforming signal hipfftExecC2C CPU \n");
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	hipfftHandle plan;
	hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C);
	hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
	hipMemcpy(h_signal_fft, d_signal, NX * NY * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	fftshift((hipfftComplex *)h_signal_fft, (hipfftComplex *)h_shift_signal_fft, NX, NY);
	padding((hipfftComplex *)h_shift_signal_fft, (hipfftComplex *)h_padding, NX, NY, NX_RES, NY_RES);

	ifftshift((hipfftComplex *)h_padding, (hipfftComplex *)h_shift_signal_ifft, NX_RES, NY_RES);

	hipMemcpy(d_signal_t, h_shift_signal_ifft, NX_RES * NY_RES * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftPlan2d(&plan, NX_RES, NY_RES, HIPFFT_C2C);
	hipfftExecC2C(plan, (hipfftComplex *)d_signal_t, (hipfftComplex *)d_signal_t, HIPFFT_BACKWARD);
	hipMemcpy(h_reversed_signal, d_signal_t, NX_RES * NY_RES * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	time = (float)duration_cast<microseconds>(t2 - t1).count();
	free(h_signal_ifft);
	free(h_shift_signal_ifft);
	free(h_signal_fft);
	free(h_shift_signal_fft);
	hipFree(d_signal);
	hipFree(d_signal_t);
	hipfftDestroy(plan);
}

void calcFFTGPU(hipfftComplex *h_signal, hipfftComplex *h_padding, hipfftComplex *h_reversed_signal, int NX, int NY, int NX_RES, int NY_RES, float &time)
{
	hipfftComplex *d_signal, *d_signal_shift, *d_signal_padding, *d_signal_shift_ifft;
	hipMalloc((void **)&d_signal, NX * NY * sizeof(hipfftComplex));
	hipMalloc((void **)&d_signal_shift, NX * NY * sizeof(hipfftComplex));
	hipMalloc((void **)&d_signal_padding, NX_RES * NY_RES * sizeof(hipfftComplex));
	hipMalloc((void **)&d_signal_shift_ifft, NX_RES * NY_RES * sizeof(hipfftComplex));
	hipSetDevice(0);
	// Copy host memory to device
	hipMemcpy(d_signal, h_signal, NX * NY * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_signal_padding, h_padding, NX_RES*NY_RES * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftHandle plan;
	hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C);

	// Transform signal and kernel
	printf("Transforming signal hipfftExecC2C\n");
	hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);

	dim3 blockDim(32, 32, 1);
	dim3 gridDim((NX + 31) / 32, (NY + 31) / 32, 1);
	dim3 gridDim2((NX_RES + 31) / 32, (NY_RES + 31) / 32, 1);
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	circshiftGPU << <gridDim, blockDim >> > ((hipfftComplex *)d_signal, (hipfftComplex *)d_signal_shift, NX, NY, (NX / 2), (NY / 2));
	paddingGPU << < gridDim, blockDim >> > ((hipfftComplex *)d_signal_shift, (hipfftComplex *)d_signal_padding, NX, NY, NX_RES, NY_RES);
	circshiftGPU << <gridDim2, blockDim >> > ((hipfftComplex *)d_signal_padding, (hipfftComplex *)d_signal_shift_ifft, NX_RES, NY_RES, ((NX_RES + 1) / 2), ((NY_RES + 1) / 2));
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	time = (float)duration_cast<microseconds>(t2 - t1).count();
	printf("Transforming signal back hipfftExecC2C\n");
	hipfftPlan2d(&plan, NX_RES, NY_RES, HIPFFT_C2C);
	hipfftExecC2C(plan, (hipfftComplex *)d_signal_shift_ifft, (hipfftComplex *)d_signal_shift_ifft, HIPFFT_BACKWARD);

	// Copy device to host memory
	hipMemcpy(h_reversed_signal, d_signal_shift_ifft, NX_RES * NY_RES * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	hipFree(d_signal);
	hipFree(d_signal_padding);
	hipFree(d_signal_shift);
	hipFree(d_signal_shift_ifft);
	hipfftDestroy(plan);
}

int main()
{
	int NX, NY, NX_RES, NY_RES;
	float factory, time;
	char choice;
	hipfftComplex *h_signal, *h_reversed_signal, *h_padding;
	Mat img, imgResize;
	img = imread("lena2.jpg", IMREAD_COLOR);

	cvtColor(img, img, cv::COLOR_BGR2GRAY);
	img.convertTo(img, CV_64F, 1.0 / 255.0);
	NX = img.cols;
	NY = img.rows;
	while (true)
	{
		cout << "Factory: ";
		cin >> factory;

		if (factory <= 0.0)
		{
			cout << "Wrong input." << endl;
			system("pause");
			return 0;
		}

		cout << "Which architecture(g/c): ";
		cin >> choice;

		if (choice != 'g' && choice != 'c')
		{
			cout << "Wrong input." << endl;;
			system("pause");
			return 0;
		}

		namedWindow("Source window", WINDOW_AUTOSIZE);
		imshow("Source window", img);
		waitKey(1);

		NX_RES = (int)(NX * factory);
		NY_RES = (int)(NY * factory);
		resize(img, imgResize, Size(), factory, factory);

		h_signal = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX * NY);
		h_reversed_signal = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX_RES * NY_RES);
		h_padding = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX_RES * NY_RES);

		for (unsigned int i = 0; i < NX*NY; i++)
			h_signal[i].x = img.at<double>((int)(i / NX), i%NX);

		for (unsigned int i = 0; i < NX_RES*NY_RES; i++)
		{
			h_padding[i].x = 0;
			h_padding[i].y = 0;
		}

		if (choice == 'g')
		{
			calcFFTGPU(h_signal, h_padding, h_reversed_signal, NX, NY, NX_RES, NY_RES, time);
		}
		else if (choice == 'c')
		{
			calcFFTCPU(h_signal, h_padding, h_reversed_signal, NX, NY, NX_RES, NY_RES, time);
		}

		cout << "Time needed to calculate FFT: " << time << " us." << endl;


		// check result
		for (unsigned int i = 0; i < NX_RES * NY_RES; i++)
		{
			h_reversed_signal[i].x = h_reversed_signal[i].x / (double)(NX_RES*NY_RES);
			h_reversed_signal[i].y = h_reversed_signal[i].y / (double)(NX_RES*NY_RES);
		}

		Mat result = Mat::zeros(NY_RES, NX_RES, CV_64F);
		// Initalize the memory for the signal
		for (unsigned int i = 0; i < NX_RES*NY_RES; i++)
			result.at<double>((int)(i / NX_RES), i%NX_RES) = h_reversed_signal[i].x;

		normalize(result, result, 1, 0, NORM_INF);

		result.convertTo(result, CV_32F);
		//medianBlur(result, result, 3);

		namedWindow("Result window", WINDOW_AUTOSIZE);
		imshow("Result window", result);
		result.convertTo(result, CV_8UC3, 255);
		imwrite("lenaFFTMedian.jpg", result);
		waitKey(1);

		namedWindow("Resized OpenCV", WINDOW_AUTOSIZE);
		imshow("Resized OpenCV", imgResize);
		waitKey(0);

		destroyWindow("Resized OpenCV");
		destroyWindow("Source window");
		destroyWindow("Result window");
		result.release();
		imgResize.release();

		// cleanup memory
		free(h_signal);
		free(h_padding);
		free(h_reversed_signal);
		hipDeviceReset();
	}

	img.release();
	system("pause");
	return 0;
}
